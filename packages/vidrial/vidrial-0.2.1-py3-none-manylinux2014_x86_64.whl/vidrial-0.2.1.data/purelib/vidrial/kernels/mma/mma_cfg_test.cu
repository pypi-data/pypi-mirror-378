#include <gtest/gtest.h>
#include <hip/hip_runtime.h>
#include "mma_cfg.cuh"
#include "mma/kernel.cuh"
#include "ABC_utils.cuh"
#include "perf_cfg.cuh"

using namespace cute;

namespace vidrial {
namespace {

TEST(MmaCfgTest, SimpleFmaMmaTest) {
    using Atom = MMA_Atom<UniversalFMA<half_t, half_t, half_t, half_t>>;
    using MNKPTestShape = Shape<_1,_8,_16,_1>;
    using MNKTileShape = Shape<_1,_8,_1>;
    using MNKAtomPlacement = Shape<_1,_4,_1>;
    auto AShape = ABC_get_MNKP(A_t{}, MNKPTestShape{});
    auto BShape = ABC_get_MNKP(B_t{}, MNKPTestShape{});
    auto CShape = ABC_get_MNKP(C_t{}, MNKPTestShape{});
    auto gA = make_managed_tensor<half_t>(make_layout(AShape));
    auto gB = make_managed_tensor<half_t>(make_layout(BShape));
    auto gC = make_managed_tensor<half_t>(make_layout(CShape));
    for (int i = 0; i < size(gA); ++i) gA(i) = static_cast<half_t>(i%14/14.);
    for (int i = 0; i < size(gB); ++i) gB(i) = static_cast<half_t>(i%27/27.);
    auto cfg = MmaKernelCfg<half_t, Atom, MNKTileShape, MNKAtomPlacement, decltype(gA.layout()), decltype(gB.layout()), decltype(gC.layout()), DefaultPerfCfg>{};
    launch_tiled_mma_kernel(cfg, gA.data(), gB.data(), gC.data());
    hipDeviceSynchronize(); hipError_t error = hipGetLastError();
    if (error != hipSuccess) { std::cerr << "CUDA error: " << hipGetErrorString(error) << std::endl; }
    auto gC_ref = make_managed_tensor<half_t>(gC.layout());
    clear(gC_ref);
    for (int p=0; p<int(get<3>(MNKPTestShape{})); ++p)
        gemm(gA(_,_,p), gB(_,_,p), gC_ref(_,_,p));
    bool match = check_tensors_match(gC, gC_ref, 1e-1, false);
    EXPECT_TRUE(match);
}

TEST(MmaCfgTest, SimpleMmaTest) {
    using MNKPTestShape = Shape<_64,_64,_128,_1>;
    using MNKTileShape = Shape<_32,_32,_16>;
    auto atom = default_MMA_atom<half_t>();
    using MNKAtomPlacement = Shape<_2,_2,_1>;
    auto AShape = ABC_get_MNKP(A_t{}, MNKPTestShape{});
    auto BShape = ABC_get_MNKP(B_t{}, MNKPTestShape{});
    auto CShape = ABC_get_MNKP(C_t{}, MNKPTestShape{});
    auto gA = make_managed_tensor<half_t>(make_layout(AShape));
    auto gB = make_managed_tensor<half_t>(make_layout(BShape));
    auto gC = make_managed_tensor<half_t>(make_layout(CShape));
    for (int i = 0; i < size(gA); ++i) gA(i) = static_cast<half_t>(i%14/14.);
    for (int i = 0; i < size(gB); ++i) gB(i) = static_cast<half_t>(i%27/27.);
    auto cfg = make_mma_cfg<half_t>(MNKTileShape{}, atom, MNKAtomPlacement{}, gA.layout(), gB.layout(), gC.layout());
    launch_tiled_mma_kernel(cfg, gA.data(), gB.data(), gC.data());
    hipDeviceSynchronize(); hipError_t error = hipGetLastError();
    if (error != hipSuccess) { std::cerr << "CUDA error: " << hipGetErrorString(error) << std::endl; }
    auto gC_ref = make_managed_tensor<half_t>(gC.layout());
    clear(gC_ref);
    for (int p=0; p<int(get<3>(MNKPTestShape{})); ++p)
        gemm(gA(_,_,p), gB(_,_,p), gC_ref(_,_,p));
    bool match = check_tensors_match(gC, gC_ref, 1e-1, false);
    EXPECT_TRUE(match);
}

template<bool AColMaj, bool BColMaj, bool CColMaj, typename MNKTileShape, typename MNKAtomPlacement, typename T>
void test_mma_kernel(auto AShape, auto BShape, auto CShape, T* A_ptr, T* B_ptr, T* C_ptr) {
    // If the ABC matrices are row major we don't want the P (batch) dimension to go first
    auto ARowMaj = select<1,0,2>(make_layout(select<1,0,2>(AShape)));
    auto BRowMaj = select<1,0,2>(make_layout(select<1,0,2>(BShape)));
    auto CRowMaj = select<1,0,2>(make_layout(select<1,0,2>(CShape)));
    auto A = std::conditional_t<AColMaj, Layout<decltype(AShape)>, decltype(ARowMaj)>{};
    auto B = std::conditional_t<BColMaj, Layout<decltype(BShape)>, decltype(BRowMaj)>{};
    auto C = std::conditional_t<CColMaj, Layout<decltype(CShape)>, decltype(CRowMaj)>{};
    auto atom = default_MMA_atom<T>();
    auto cfg = make_mma_cfg<T>(MNKTileShape{}, atom, MNKAtomPlacement{}, A, B, C);
    launch_tiled_mma_kernel(cfg, A_ptr, B_ptr, C_ptr);
    hipDeviceSynchronize(); hipError_t error = hipGetLastError();
    if (error != hipSuccess) { std::cerr << "CUDA error: " << hipGetErrorString(error) << std::endl; }
    auto gA = make_tensor(A_ptr, cfg.A.gSlab);
    auto gB = make_tensor(B_ptr, cfg.B.gSlab);
    auto gC = make_tensor(C_ptr, cfg.C.gSlab);
    default_sTile(select<0,1>(cfg.C.gSlab), cfg.C.sTile.shape());
    EXPECT_EQ(error, hipSuccess);
    auto gC_ref = make_managed_tensor<half_t>(gC.layout());
    clear(gC_ref);
    for (int p=0; p<cfg.P; ++p)
        gemm(gA(_,_,p), gB(_,_,p), gC_ref(_,_,p));
    bool match = check_tensors_match(gC, gC_ref, 1e-1, false);
    EXPECT_TRUE(match);
}
template<typename MNKPTestShape, typename MNKTileShape, typename MNKAtomPlacement>
void test_default_mma_kernel_all_rowcol_combinations() {
    auto AShape = ABC_get_MNKP(A_t{}, MNKPTestShape{});
    auto BShape = ABC_get_MNKP(B_t{}, MNKPTestShape{});
    auto CShape = ABC_get_MNKP(C_t{}, MNKPTestShape{});
    auto _gA = make_managed_tensor<half_t>(make_layout(AShape));
    auto _gB = make_managed_tensor<half_t>(make_layout(BShape));
    auto _gC = make_managed_tensor<half_t>(make_layout(CShape));
    for (int i = 0; i < size(_gA); ++i) _gA(i) = static_cast<half_t>(i%14/14.);
    for (int i = 0; i < size(_gB); ++i) _gB(i) = static_cast<half_t>(i%27/27.);
    auto A_ptr = _gA.data();
    auto B_ptr = _gB.data();
    auto C_ptr = _gC.data();
    // only try some combinations to speed up the test
    test_mma_kernel<1,1,1, MNKTileShape, MNKAtomPlacement>(AShape, BShape, CShape, A_ptr, B_ptr, C_ptr);
    // test_mma_kernel<1,1,0, MNKTileShape, MNKAtomPlacement>(AShape, BShape, CShape, A_ptr, B_ptr, C_ptr);
    test_mma_kernel<1,0,1, MNKTileShape, MNKAtomPlacement>(AShape, BShape, CShape, A_ptr, B_ptr, C_ptr);
    // test_mma_kernel<1,0,0, MNKTileShape, MNKAtomPlacement>(AShape, BShape, CShape, A_ptr, B_ptr, C_ptr);
    // test_mma_kernel<0,1,1, MNKTileShape, MNKAtomPlacement>(AShape, BShape, CShape, A_ptr, B_ptr, C_ptr);
    // test_mma_kernel<0,1,0, MNKTileShape, MNKAtomPlacement>(AShape, BShape, CShape, A_ptr, B_ptr, C_ptr);
    test_mma_kernel<0,0,1, MNKTileShape, MNKAtomPlacement>(AShape, BShape, CShape, A_ptr, B_ptr, C_ptr);
    test_mma_kernel<0,0,0, MNKTileShape, MNKAtomPlacement>(AShape, BShape, CShape, A_ptr, B_ptr, C_ptr);
}
TEST(MmaCfgTest, mma_kernel_Shape16x16x16_Tile16x16x16_AtomPlacement1x1x1) {
    using MNKPTestShape = Shape<_16,_16,_16,_1>;
    using MNKTileShape = Shape<_16,_16,_16>;
    using MNKAtomPlacement = Shape<_1,_1,_1>;
    test_default_mma_kernel_all_rowcol_combinations<MNKPTestShape, MNKTileShape, MNKAtomPlacement>();
}
TEST(MmaCfgTest, mma_kernel_Shape16x16x16_Tile16x16x16_AtomPlacement1x2x1) {
    using MNKPTestShape = Shape<_16,_16,_16,_1>;
    using MNKTileShape = Shape<_16,_16,_16>;
    using MNKAtomPlacement = Shape<_1,_2,_1>;
    test_default_mma_kernel_all_rowcol_combinations<MNKPTestShape, MNKTileShape, MNKAtomPlacement>();
}
TEST(MmaCfgTest, mma_kernel_Shape64x128x64_Tile16x8x16) {
    using MNKPTestShape = Shape<_64,_128,_64,_4>;
    using MNKTileShape = Shape<_64,_64,_16>;
    using MNKAtomPlacement = Shape<_2,_4,_1>;
    test_default_mma_kernel_all_rowcol_combinations<MNKPTestShape, MNKTileShape, MNKAtomPlacement>();
}
TEST(MmaCfgTest, mma_kernel_Shape64x128x64_Tile32x32x32) {
    using MNKPTestShape = Shape<_64,_128,_64,_2>;
    using MNKTileShape = Shape<_32,_32,_32>;
    using MNKAtomPlacement = Shape<_2,_2,_1>;
    test_default_mma_kernel_all_rowcol_combinations<MNKPTestShape, MNKTileShape, MNKAtomPlacement>();
}
} // namespace
} // namespace vidrial 